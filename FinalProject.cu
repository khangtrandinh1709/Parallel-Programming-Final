#include <iostream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

// Hyperparameters
#define INPUT_SIZE 784
#define HIDDEN_LAYER_1 128
#define HIDDEN_LAYER_2 128
#define OUTPUT_SIZE 10
#define BATCH_SIZE 32
#define LEARNING_RATE 0.01
#define EPOCHS 1

// Activation Functions
__device__ float relu(float x) {
    return x > 0 ? x : 0;
}

__device__ float relu_derivative(float x) {
    return x > 0 ? 1 : 0;
}

__device__ void softmax(float *output, int size) {
    float max_val = -1e9;
    for (int i = 0; i < size; ++i) max_val = max(max_val, output[i]);

    float sum = 0.0;
    for (int i = 0; i < size; ++i) {
        output[i] = expf(output[i] - max_val);
        sum += output[i];
    }

    for (int i = 0; i < size; ++i) output[i] /= sum;
}

// GPU Kernel for Forward Propagation
__global__ void forward_pass(float *input, float *w1, float *b1, float *w2, float *b2, float *w3, float *b3, float *output, int batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) return;

    float hidden1[HIDDEN_LAYER_1] = {0};
    float hidden2[HIDDEN_LAYER_2] = {0};

    // Layer 1 - Input to Hidden 1
    for (int i = 0; i < HIDDEN_LAYER_1; ++i) {
        hidden1[i] = b1[i];
        for (int j = 0; j < INPUT_SIZE; ++j) {
            hidden1[i] += input[idx * INPUT_SIZE + j] * w1[i * INPUT_SIZE + j];
        }
        hidden1[i] = relu(hidden1[i]);
    }

    // Layer 2 - Hidden 1 to Hidden 2
    for (int i = 0; i < HIDDEN_LAYER_2; ++i) {
        hidden2[i] = b2[i];
        for (int j = 0; j < HIDDEN_LAYER_1; ++j) {
            hidden2[i] += hidden1[j] * w2[i * HIDDEN_LAYER_1 + j];
        }
        hidden2[i] = relu(hidden2[i]);
    }

    // Layer 3 - Hidden 2 to Output
    for (int i = 0; i < OUTPUT_SIZE; ++i) {
        output[idx * OUTPUT_SIZE + i] = b3[i];
        for (int j = 0; j < HIDDEN_LAYER_2; ++j) {
            output[idx * OUTPUT_SIZE + i] += hidden2[j] * w3[i * HIDDEN_LAYER_2 + j];
        }
    }

    // Softmax
    softmax(&output[idx * OUTPUT_SIZE], OUTPUT_SIZE);
}

int main() {
    // Allocate memory for weights, biases, inputs, and outputs
    float *h_input, *h_w1, *h_b1, *h_w2, *h_b2, *h_w3, *h_b3, *h_output;
    float *d_input, *d_w1, *d_b1, *d_w2, *d_b2, *d_w3, *d_b3, *d_output;

    h_input = (float *)malloc(BATCH_SIZE * INPUT_SIZE * sizeof(float));
    h_w1 = (float *)malloc(HIDDEN_LAYER_1 * INPUT_SIZE * sizeof(float));
    h_b1 = (float *)malloc(HIDDEN_LAYER_1 * sizeof(float));
    h_w2 = (float *)malloc(HIDDEN_LAYER_2 * HIDDEN_LAYER_1 * sizeof(float));
    h_b2 = (float *)malloc(HIDDEN_LAYER_2 * sizeof(float));
    h_w3 = (float *)malloc(OUTPUT_SIZE * HIDDEN_LAYER_2 * sizeof(float));
    h_b3 = (float *)malloc(OUTPUT_SIZE * sizeof(float));
    h_output = (float *)malloc(BATCH_SIZE * OUTPUT_SIZE * sizeof(float));

    hipMalloc(&d_input, BATCH_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&d_w1, HIDDEN_LAYER_1 * INPUT_SIZE * sizeof(float));
    hipMalloc(&d_b1, HIDDEN_LAYER_1 * sizeof(float));
    hipMalloc(&d_w2, HIDDEN_LAYER_2 * HIDDEN_LAYER_1 * sizeof(float));
    hipMalloc(&d_b2, HIDDEN_LAYER_2 * sizeof(float));
    hipMalloc(&d_w3, OUTPUT_SIZE * HIDDEN_LAYER_2 * sizeof(float));
    hipMalloc(&d_b3, OUTPUT_SIZE * sizeof(float));
    hipMalloc(&d_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float));

    // Initialize weights and biases randomly
    for (int i = 0; i < HIDDEN_LAYER_1 * INPUT_SIZE; ++i) h_w1[i] = ((float)rand() / RAND_MAX) * 0.01;
    for (int i = 0; i < HIDDEN_LAYER_1; ++i) h_b1[i] = 0.0;

    for (int i = 0; i < HIDDEN_LAYER_2 * HIDDEN_LAYER_1; ++i) h_w2[i] = ((float)rand() / RAND_MAX) * 0.01;
    for (int i = 0; i < HIDDEN_LAYER_2; ++i) h_b2[i] = 0.0;

    for (int i = 0; i < OUTPUT_SIZE * HIDDEN_LAYER_2; ++i) h_w3[i] = ((float)rand() / RAND_MAX) * 0.01;
    for (int i = 0; i < OUTPUT_SIZE; ++i) h_b3[i] = 0.0;

    // Copy data to device
    hipMemcpy(d_input, h_input, BATCH_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w1, h_w1, HIDDEN_LAYER_1 * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, h_b1, HIDDEN_LAYER_1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w2, h_w2, HIDDEN_LAYER_2 * HIDDEN_LAYER_1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, h_b2, HIDDEN_LAYER_2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w3, h_w3, OUTPUT_SIZE * HIDDEN_LAYER_2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, h_b3, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Forward Pass
    forward_pass<<<(BATCH_SIZE + 31) / 32, 32>>>(d_input, d_w1, d_b1, d_w2, d_b2, d_w3, d_b3, d_output, BATCH_SIZE);

    // Copy output back to host
    hipMemcpy(h_output, d_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    free(h_input); free(h_w1); free(h_b1); free(h_w2); free(h_b2); free(h_w3); free(h_b3); free(h_output);
    hipFree(d_input); hipFree(d_w1); hipFree(d_b1); hipFree(d_w2); hipFree(d_b2); hipFree(d_w3); hipFree(d_b3); hipFree(d_output);

    std::cout << "Forward pass completed!" << std::endl;

    return 0;
}
